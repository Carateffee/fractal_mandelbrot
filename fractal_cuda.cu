#include <stdint.h>

class Bmp256 {										//自定义图像类
#pragma pack(2) 									// 设定变量以n = 2字节对齐方式
	struct Header {									// 头信息
		uint16_t bfType = 0x4D42;
		uint32_t bfSize;
		uint16_t bfReserved1 = 0;
		uint16_t bfReserved2 = 0;
		uint32_t bfOffBits = 54 + 256 * 4;
		uint32_t biSize = 40;
		int32_t  biWidth;
		int32_t  biHeight;
		uint16_t biPlanes = 1;
		uint16_t biBitCount = 8;
		uint32_t biCompression = 0;
		uint32_t biSizeImage = 0;
		int32_t  biXPelsPerMeter = 0;
		int32_t  biYPelsPerMeter = 0;
		uint32_t biClrUsed = 256;
		uint32_t biClrImportant = 256;
	} header;
#pragma pack()										// 默认值8
	int32_t rowSize;								// 行大小
	struct {										// 像素点通道结构
		uint8_t B, G, R, A;
		void set(uint8_t r, uint8_t g, uint8_t b) { R = r; G = g; B = b; A = 0; }	// 设置颜色函数
	} palette[256];									// 256个颜色的调色板
	uint8_t *buffer = NULL;						// 图像缓存

	void calc_palette();

public:
	Bmp256(int width, int height);					// 类构造函数
	~Bmp256() { delete[] buffer; }					// 类析构函数
	int width()  const { return header.biWidth; }	// 获取图像宽度
	int height() const { return header.biHeight; }	// 获取图像高度
	uint8_t& operator()(int row, int col) { return buffer[row * rowSize + col]; }	// get/set the pixel
	void save(const char* file);					// 保存图像
	uint8_t* get_ptr() { return buffer; };				//获取像素
	int image_size() { return header.bfSize - header.bfOffBits; };
};

Bmp256::Bmp256(int width, int height) {
	header.biWidth = width;							// 从头信息中获取宽度和高度
	header.biHeight = height;
	rowSize = width;				// 计算行大小
	int buffSize = rowSize * height;				// 图像整体缓存的大小
	header.bfSize = header.bfOffBits + buffSize;	
	calc_palette();									// 初始化调色板颜色
	buffer = new uint8_t[buffSize];					// 新建图像缓存
}

void Bmp256::calc_palette() {
	for (int i = 0; i < 64; ++i) {
		palette[i].set(255, 255 - i * 4, 0);
		palette[i + 64].set(255 - i * 2, 0, i * 2);
		palette[i + 128].set(127 - i * 2, 0, 128 + i * 2);
		palette[i + 192].set(0, 0, 255 - i * 3);
	}
	palette[0].set(0, 0, 0);
}

#include <iostream>
#include <fstream>
void Bmp256::save(const char* file_name) {			// 保存
	std::ofstream of(file_name, std::ios::binary);
	of.write((char *)&header, 54);
	of.write((char *)palette, 256 * 4);
	char* p = (char *)buffer;
	for (int i = 0; i < header.biHeight; ++i) {
		of.write(p, rowSize);
		p += rowSize;
	}
}



const double RMIN = -2, RMAX = 1, IMIN = -1, IMAX = 1;// 实部和虚部的范围
const int W = 12 * 1024;							// 宽度：12*1024
const double RESN = W / (RMAX - RMIN);				// 实部单位像素数12*1024/（1-(-2)）=4*1024
const int H = (IMAX - IMIN) * RESN;					// 高度：（1-(-1)）*4*1024=8*1024
const int MI = 1;

// int Mandelbrot(complex c) {							// 曼德博集合是一种在复平面上组成分形的点的集合
// 	complex z;
// 	for (int k = 256 * MI - 1; k >= 0; --k) {
// 		z = z * z + c;
// 		if (std::norm(z) > 4) return k / MI;		//计算分形
// 	}
// 	return 0;
// }


#include <ctime>
#include <hip/hip_runtime.h>

struct hipComplex {
	float r;
	float i;
	__device__ hipComplex( float a, float b ) : r(a), i(b) {}
	__device__ float magnitude2( void ) {
		return r * r + i * i;
	}
	__device__ hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ hipComplex operator+(const hipComplex& a) {
		return hipComplex(r+a.r, i+a.i);
	}
};

__device__ int Mandelbrot( int x, int y ) {	
	float jx = RMIN + x / RESN;
	float jy = IMIN + y / RESN;

	hipComplex a(0, 0);
	hipComplex c(jx, jy);
	int k = 256 * MI - 1;
	for (int k = 256 * MI - 1; k >= 0; --k) {
		a = a * a + c;
        if (a.magnitude2() > 4) {return k / MI;};
	}
	return 0;
}

__global__ void kernel( uint8_t *ptr ) {
	// map from threadIdx/BlockIdx to position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * W;

	ptr[offset] =  Mandelbrot( x, y );

}

int main() {
	Bmp256 bmp(W, H);

	clock_t t1 = clock();;					// 单调计时时钟
	uint8_t *dev_bitmap;

	hipMalloc( (void**)&dev_bitmap,
							 bmp.image_size());

	dim3 grid(W, H);
	kernel<<<grid,1>>>( dev_bitmap );

	hipMemcpy( bmp.get_ptr(),
							  dev_bitmap,
							  bmp.image_size(),
							  hipMemcpyDeviceToHost );

	hipFree( dev_bitmap );

	clock_t t2 = clock();;

	bmp.save("Mandelbrot12k.bmp");
	std::cout << "run time: " << (double)(t2 - t1) / CLOCKS_PER_SEC << " seconds.\n";
}
